
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define ROW 512
#define N (ROW*ROW) //2048*2048
#define THREADS_PER_BLOCK (ROW) //1024


// forward declaration
__global__ void td(int *in, int *out, int * d_flag);
__global__ void soma(int *in, int *out, int * end);
__global__ void copy(int *in, int *out, int * end);
__global__ void fill(int *in, int * end);

void random_ints(int * mat, int n) {
  srand(time(0));
  int i;
  for (i = 0; i < n; i++) {
    mat[i] = rand() % 2;
  }
}

void prandom_ints(int * mat, int n) {
  int i, j;
  for (i = 0; i < ROW; i++) {
    for (j = 0; j < ROW; j++) {
      if(i == 0 || j == 0 || j + 1 == ROW || i + 1 == ROW) {
        mat[i * ROW + j] = 0;
      } else {
        mat[i * ROW + j] = 1;
      }
      
    }
  }
}

// gera numeros binarios aleatorios para a matrix
void generateRandom(int ** mat, int row, int col) {
  srand((unsigned int)time(0));
  srand(rand());
  int i, j;
  for(i = 0; i < row; i++) {
    for(j = 0; j < col; j++) {
      if(i == 0 || j == 0 || j + 1 == col || i + 1 == row) {
        mat[i][j] = 0;
      } else {
        mat[i][j] = 1;
      }
    }
  }
  mat[row - 1][j - 2] = 1;
  mat[row - 2][j - 1] = 1;
}

void printMatrix(int * mat, int n) {
  int i, j;
  for (i = 0; i < ROW; i++) {
    for (j = 0; j < ROW; j++) {
      printf("%d ", mat[i * ROW + j]);
    }
    printf("\n");
  }

}


int main(void) {
  int *in, *out, *end, * flag;     // host copies a,b
  int *d_in,*d_out, *d_end, * d_flag;  // device copies a,b
  int size = N * sizeof(int);

  // Alloc space for host copies a,b and setup input
  in  = (int *)malloc(size);
  // fill input matrix (as an example)
  prandom_ints(in, N);
  out = (int *)malloc(size);
  end = (int *)malloc(size);
  flag = (int *)malloc(sizeof(int));

  printf("IN\n");
  //printMatrix(in, N);

  // Allocate space for device copies of in, out
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);
  hipMalloc((void **)&d_end, size);
  hipMalloc((void **)&d_flag, sizeof(int));

  // copy input matrix an flag from host to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  hipMemcpy(d_flag, flag, sizeof(int), hipMemcpyHostToDevice);

  // set as true
  flag[0] = 1;
  // copy input to end
  fill<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_end);

  // enquanto tiver
  while(flag[0]) {
    td<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_out, d_flag);
    copy<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in, d_out, d_end);
    soma<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in, d_out, d_end);
    hipMemcpy(flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
  }

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
  hipMemcpy(end, d_end, size, hipMemcpyDeviceToHost);

  printf("flag: %d\n", flag[0]);

  printf("END\n");
  //printMatrix(end, N);

  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out); hipFree(d_end);
  return 0;
}

__global__ void td(int *in, int *out, int * d_flag) {
  d_flag[0] = 0;
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  int * local_in  = (int *)malloc(ROW * ROW);

  int l, m;
  float mini = ROW * ROW;
  for(l = -1; l <= 1; l++) {
    for(m = -1; m <= 1; m++) {
      // borda
      if(((int) blockIdx.x + l) >= 0 && ((int) blockIdx.x + l) < ROW && ((int) threadIdx.x + m) >= 0 && ((int) threadIdx.x + m) < ROW) {
        mini = fmin(in[(blockIdx.x + l) * blockDim.x + (threadIdx.x + m)], mini);
      }
    }
  }

  if(mini != ROW * ROW) {
    out[current_index] = mini;
  }

  if(mini != 0) {
    d_flag[0] = 1;
  }
  
}

__global__ void copy(int *in, int *out, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  in[current_index] = out[current_index];
}

__global__ void soma(int *in, int *out, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  end[current_index] += out[current_index];
}

__global__ void fill(int *in, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  end[current_index] = in[current_index];
}
