
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define ROW 16
#define N (ROW*ROW) //2048*2048
#define THREADS_PER_BLOCK (ROW) //1024


// forward declaration
__global__ void td(int *in, int *out);
__global__ void soma(int *in, int *out, int * end);


void random_ints(int * mat, int n) {
  srand(time(0));
  int i;
  for (i = 0; i < n; i++) {
    mat[i] = rand() % 2;
  }
}

void prandom_ints(int * mat, int n) {
  int i, j;
  for (i = 0; i < ROW; i++) {
    for (j = 0; j < ROW; j++) {
      if(i == 0 || j == 0 || j + 1 == ROW || i + 1 == ROW) {
        mat[i * ROW + j] = 0;
      } else {
        mat[i * ROW + j] = 1;
      }
      
    }
  }
}

// gera numeros binarios aleatorios para a matrix
void generateRandom(int ** mat, int row, int col) {
  srand((unsigned int)time(0));
  srand(rand());
  int i, j;
  for(i = 0; i < row; i++) {
    for(j = 0; j < col; j++) {
      if(i == 0 || j == 0 || j + 1 == col || i + 1 == row) {
        mat[i][j] = 0;
      } else {
        mat[i][j] = 1;
      }
    }
  }
  mat[row - 1][j - 2] = 1;
  mat[row - 2][j - 1] = 1;
}

void printMatrix(int * mat, int n) {
  int i, j;
  for (i = 0; i < ROW; i++) {
    for (j = 0; j < ROW; j++) {
      printf("%d ", mat[i * ROW + j]);
    }
    printf("\n");
  }

}


int main(void) {
  int *in, *out;     // host copies a,b
  int *d_in,*d_out, *d_end;  // device copies a,b
  int size = N * sizeof(int);

  // Alloc space for host copies a,b and setup input
  in  = (int *)malloc(size);

  prandom_ints(in, N);
  out = (int *)malloc(size);

  printf("IN\n");
  printMatrix(in, N);

  // Allocate space for device copies of in, out
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);
  hipMalloc((void **)&d_end, size);

  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

  td<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_out);
  soma<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_out,d_end);
  td<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_out);
  soma<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_out,d_end);

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
  hipMemcpy(out, d_end, size, hipMemcpyDeviceToHost);

  printf("OUT\n");
  printMatrix(out, N);
  printf("END\n");
  printMatrix(out, N);

  // Cleanupxs
  free(in); free(out);
  hipFree(d_in); hipFree(d_out); hipFree(d_end);
  return 0;
}

__device__ int barrier = N;

__global__ void td(int *in, int *out) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  int * local_in  = (int *)malloc(ROW * ROW);

  int l, m;
  float mini = ROW * ROW;
  for(l = -1; l <= 1; l++) {
    for(m = -1; m <= 1; m++) {
      // borda
      if(((int) blockIdx.x + l) >= 0 && ((int) blockIdx.x + l) < ROW && ((int) threadIdx.x + m) >= 0 && ((int) threadIdx.x + m) < ROW) {
        mini = fmin(in[(blockIdx.x + l) * blockDim.x + (threadIdx.x + m)], mini);
      }
    }
  }

  if(mini != ROW * ROW) {
    out[current_index] = mini;
  }
  
}

__global__ void soma(int *in, int *out, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  in[current_index] = out[current_index];
  end[current_index] += out[current_index];
}