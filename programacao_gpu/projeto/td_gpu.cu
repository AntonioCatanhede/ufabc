
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define ROW 2048
#define N (ROW*ROW) //2048*2048
#define THREADS_PER_BLOCK (ROW) //1024

// forward declaration
__global__ void td(int *in, int *out, int * d_flag);
__global__ void td_shared(int *in, int *out, int * d_flag);
__global__ void soma(int *in, int *out, int * end);
__global__ void copy(int *in, int *out, int * end);
__global__ void fill(int *in, int * end);

int * td_gpu(int * in, int type) {
  int *out, *end, * flag;     // host copies a,b
  int *d_in,*d_out, *d_end, * d_flag;  // device copies a,b
  int size = N * sizeof(int);

  // allocate outup space
  out = (int *)malloc(size);
  end = (int *)malloc(size);
  flag = (int *)malloc(sizeof(int));

  // Allocate space for device copies of in, out
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);
  hipMalloc((void **)&d_end, size);
  hipMalloc((void **)&d_flag, sizeof(int));

  // copy input matrix an flag from host to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

  // set as true
  flag[0] = 1;
  // copy input to end
  fill<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_end);

  // enquanto tiver
  while(flag[0]) {
    flag[0] = 0;
    hipMemcpy(d_flag, flag, sizeof(int), hipMemcpyHostToDevice);
    if(type) {
      td_shared<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_out, d_flag);
    } else {
      td<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_out, d_flag);
    }
    copy<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in, d_out, d_end);
    soma<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in, d_out, d_end);
    hipMemcpy(flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
  }

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
  hipMemcpy(end, d_end, size, hipMemcpyDeviceToHost);
  // Cleanup
  free(out);
  hipFree(d_in); hipFree(d_out); hipFree(d_end);
  return end;
}

__global__ void td(int *in, int *out, int * d_flag) {
  int local_flag = 0;
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;

  int l, m;
  float mini = ROW * ROW;
  for(l = -1; l <= 1; l++) {
    for(m = -1; m <= 1; m++) {
      // borda
      if(((int) blockIdx.x + l) >= 0 && ((int) blockIdx.x + l) < ROW && ((int) threadIdx.x + m) >= 0 && ((int) threadIdx.x + m) < ROW) {
        mini = fmin(in[(blockIdx.x + l) * blockDim.x + (threadIdx.x + m)], mini);
      }
    }
  }

  if(mini != ROW * ROW) {
    out[current_index] = mini;
  }
  // flag para continuear ou nao
  if(mini != 0) {
    local_flag = 1;
  }
  if(d_flag[0] == 0 && local_flag == 1) {
      d_flag[0] = 1;
  }
  
}

__global__ void td_shared(int *in, int *out, int * d_flag) {
  int local_flag = 0;
  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
  int lindex = threadIdx.x;

  __shared__ int temp[ROW][3];

  // checa para ver se a primeira linha existe
  // se existe poe na primeira linha da memoria compartilhada
  if(gindex - ROW > 0) {
    temp[lindex][0] = in[gindex - ROW];
  } else {
    // se nao tiver preenche com 0 -> borda superior
    temp[lindex][0] = 1;
  }
  // a segunda linha sempre vai existir
  temp[lindex][1] = in[gindex];
  // terceira linha borda inferior
  if(gindex + ROW < ROW * ROW){
    temp[lindex][2] = in[gindex + ROW];
  } else {
    // se nao tiver preenche com 0 -> borda superior
    temp[lindex][2] = 1;
  }

  __syncthreads();

  int l, m;
  float mini = ROW * ROW;
  for(l = 0; l <= 2; l++) {
    for(m = -1; m <= 1; m++) {
      // borda
      if(lindex + m >= 0 && lindex + m < ROW) {
        mini = fmin(temp[lindex + m][l], mini);
      }
    }
  }

  if(mini != ROW * ROW) {
     out[gindex] = mini;
  }
  // flag para continuear ou nao
  if(mini != 0) {
    local_flag = 1;
  }
  if(d_flag[0] == 0 && local_flag == 1) {
      d_flag[0] = 1;
  }
  
}

__global__ void copy(int *in, int *out, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  in[current_index] = out[current_index];
}

__global__ void soma(int *in, int *out, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  end[current_index] += out[current_index];
}

__global__ void fill(int *in, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  end[current_index] = in[current_index];
}
