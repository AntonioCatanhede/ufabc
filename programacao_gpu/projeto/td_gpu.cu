
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define ROW 512
#define N (ROW*ROW) //2048*2048
#define THREADS_PER_BLOCK (ROW) //1024

// forward declaration
__global__ void td(int *in, int *out, int * d_flag);
__global__ void soma(int *in, int *out, int * end);
__global__ void copy(int *in, int *out, int * end);
__global__ void fill(int *in, int * end);

int * td_gpu(int * in) {
  float time_;
  clock_t t;
  int *out, *end, * flag;     // host copies a,b
  int *d_in,*d_out, *d_end, * d_flag;  // device copies a,b
  int size = N * sizeof(int);

  // allocate outup space
  out = (int *)malloc(size);
  end = (int *)malloc(size);
  flag = (int *)malloc(sizeof(int));

  // Allocate space for device copies of in, out
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);
  hipMalloc((void **)&d_end, size);
  hipMalloc((void **)&d_flag, sizeof(int));

  // copy input matrix an flag from host to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  hipMemcpy(d_flag, flag, sizeof(int), hipMemcpyHostToDevice);

  time_ = 0;
  t = clock();

  // set as true
  flag[0] = 1;
  // copy input to end
  fill<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_end);

  // enquanto tiver
  while(flag[0]) {
    td<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_out, d_flag);
    copy<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in, d_out, d_end);
    soma<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in, d_out, d_end);
    hipMemcpy(flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
  }

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
  hipMemcpy(end, d_end, size, hipMemcpyDeviceToHost);
  time_ = (float)(clock() - t);
  time_ = time_ / CLOCKS_PER_SEC;
  printf("Tempo GPU (apenas TD): %5.1fms\n", time_ * 1000);
  return end;
  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out); hipFree(d_end);
}

__global__ void td(int *in, int *out, int * d_flag) {
  d_flag[0] = 0;
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  int * local_in  = (int *)malloc(ROW * ROW);

  int l, m;
  float mini = ROW * ROW;
  for(l = -1; l <= 1; l++) {
    for(m = -1; m <= 1; m++) {
      // borda
      if(((int) blockIdx.x + l) >= 0 && ((int) blockIdx.x + l) < ROW && ((int) threadIdx.x + m) >= 0 && ((int) threadIdx.x + m) < ROW) {
        mini = fmin(in[(blockIdx.x + l) * blockDim.x + (threadIdx.x + m)], mini);
      }
    }
  }

  if(mini != ROW * ROW) {
    out[current_index] = mini;
  }
  // flag para continuear ou nao
  if(mini != 0) {
    d_flag[0] = 1;
  }
  
}

__global__ void copy(int *in, int *out, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  in[current_index] = out[current_index];
}

__global__ void soma(int *in, int *out, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  end[current_index] += out[current_index];
}

__global__ void fill(int *in, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  end[current_index] = in[current_index];
}
