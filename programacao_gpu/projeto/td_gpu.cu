
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define ROW 512
#define N (ROW*ROW) //2048*2048
#define THREADS_PER_BLOCK (ROW) //1024

// forward declaration
__global__ void td(int *in, int *out, int * d_flag);
__global__ void soma(int *in, int *out, int * end);
__global__ void copy(int *in, int *out, int * end);
__global__ void fill(int *in, int * end);

int * td_gpu(int * in) {
  int *out, *end, * flag;     // host copies a,b
  int *d_in,*d_out, *d_end, * d_flag;  // device copies a,b
  int size = N * sizeof(int);

  // allocate outup space
  out = (int *)malloc(size);
  end = (int *)malloc(size);
  flag = (int *)malloc(sizeof(int));

  // Allocate space for device copies of in, out
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);
  hipMalloc((void **)&d_end, size);
  hipMalloc((void **)&d_flag, sizeof(int));

  // copy input matrix an flag from host to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

  // set as true
  flag[0] = 1;
  // copy input to end
  fill<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_end);

  // enquanto tiver
  while(flag[0]) {
    flag[0] = 0;
    hipMemcpy(d_flag, flag, sizeof(int), hipMemcpyHostToDevice);
    td<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_out, d_flag);
    copy<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in, d_out, d_end);
    soma<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in, d_out, d_end);
    hipMemcpy(flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
  }

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
  hipMemcpy(end, d_end, size, hipMemcpyDeviceToHost);
  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out); hipFree(d_end);
  return end;
}

__global__ void td(int *in, int *out, int * d_flag) {
  int local_flag = 0;
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  int * local_in  = (int *)malloc(ROW * ROW);

  int l, m;
  float mini = ROW * ROW;
  for(l = -1; l <= 1; l++) {
    for(m = -1; m <= 1; m++) {
      // borda
      if(((int) blockIdx.x + l) >= 0 && ((int) blockIdx.x + l) < ROW && ((int) threadIdx.x + m) >= 0 && ((int) threadIdx.x + m) < ROW) {
        mini = fmin(in[(blockIdx.x + l) * blockDim.x + (threadIdx.x + m)], mini);
      }
    }
  }

  if(mini != ROW * ROW) {
    out[current_index] = mini;
  }
  // flag para continuear ou nao
  if(mini != 0) {
    local_flag = 1;
  }
  if(d_flag[0] == 0 && local_flag == 1) {
      d_flag[0] = 1;
  }
  
}

__global__ void copy(int *in, int *out, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  in[current_index] = out[current_index];
}

__global__ void soma(int *in, int *out, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  end[current_index] += out[current_index];
}

__global__ void fill(int *in, int * end) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  end[current_index] = in[current_index];
}
