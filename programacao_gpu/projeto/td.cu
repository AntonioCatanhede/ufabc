
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define ROW 16
#define N (ROW*ROW) //2048*2048
#define THREADS_PER_BLOCK (ROW) //1024


// forward declaration
__global__ void td(int *in, int *out);


void random_ints(int * mat, int n) {
  srand(time(0));
  int i;
  for (i = 0; i < n; i++) {
    mat[i] = rand() % 2;
  }
}

void printMatrix(int * mat, int n) {
  int i, j;
  for (i = 0; i < ROW; i++) {
    for (j = 0; j < ROW; j++) {
      printf("%d ", mat[i * ROW + j]);
    }
    printf("\n");
  }

}


int main(void) {
  int *in, *out;     // host copies a,b
  int *d_in,*d_out;  // device copies a,b
  int size = N * sizeof(int);

  // Alloc space for host copies a,b and setup input
  in  = (int *)malloc(size);

  random_ints(in, N);
  out = (int *)malloc(size);

  printf("IN\n");
  printMatrix(in, N);

  // Allocate space for device copies of in, out
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);

  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

  td<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_out);

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

  printf("OUT\n");
  printMatrix(out, N);

  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  return 0;
}



__global__ void td(int *in, int *out) {
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  int last_index = (ROW * ROW); // ultimo indice

  int distancia_esq = 0;
  int distancia_dir = 0;
  // olha todas as casas à esquerda da posição original
  for(int offset = current_index; offset >= 0; offset--) {
  	if((int)in[offset] == 0){
  		out[current_index] = distancia_esq; // achou a distancia minima a esquerda
  		break;
  	}
    // nao achou incrementa e continua
    distancia_esq++;
  }
  __syncthreads(); // espera todos à esquerda terminar

  // olha todas as casas à direita da posição original
  for(int offset = current_index; offset < last_index; offset++){
  	if((int)in[offset] == 0) {
  		// ver se a distancia a direita eh menor que a esquerda ja calculada
  		if(out[current_index] > distancia_dir) {
  			out[current_index] = distancia_dir;
  		}
  		break; // achou a distancia minima a direita
  	}
  	distancia_dir++;
  }
}
