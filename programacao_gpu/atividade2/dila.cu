
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define ROW 6
#define N (6*6) //2048*2048
#define THREADS_PER_BLOCK (ROW/2) //1024
#define RADIUS 3
#define BLOCK_SIZE (THREADS_PER_BLOCK-2*RADIUS)

// forward declaration
__global__ void dila(int *in, int *out);


void random_ints(int * mat, int n) {
  srand(time(0));
  int i;
  for (i = 0; i < n; i++) {
    mat[i] = (rand() % 2) * 10;
  }
}

void printMatrix(int * mat, int n) {

  int i, j;
  for (i = 0; i < ROW; i++) {
    for (j = 0; j < ROW; j++) {
      printf("%d ", mat[i * ROW + j]);
    }
    printf("\n");
  }

}


int main(void) {
  int *in, *out;     // host copies a,b
  int *d_in,*d_out;  // device copies a,b
  int size = N * sizeof(int);

  // Alloc space for host copies a,b and setup input
  in  = (int *)malloc(size);

  random_ints(in, N);
  out = (int *)malloc(size);

  printf("IN\n");
  printMatrix(in, N);

  // Allocate space for device copies of in, out
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);

  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);


  dila<<< (N/THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(d_in,d_out);

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

  printf("OUTZ\n");
  printMatrix(out, N);

  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  return 0;
}



__global__ void dila(int *in, int *out) {
  __shared__ float temp[BLOCK_SIZE + 2 * RADIUS];
  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
  int lindex = threadIdx.x + RADIUS;
  // 1. Read input elements into shared memory
  temp[lindex] = in[gindex];
  if (threadIdx.x < RADIUS) {
    temp[lindex - RADIUS]     = in[gindex - RADIUS];
    temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
  }
  __syncthreads();
  // 2. Apply the dilatation
  float result = 0.0; // low number
  for (int offset = -RADIUS ; offset <= RADIUS ; offset++){
	   result = fmaxf(result, temp[lindex + offset]);

  }
  // 3. Copy the result
  out[gindex] = result;
}
