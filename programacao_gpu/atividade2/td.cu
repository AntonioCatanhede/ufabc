
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define ROW 8
#define N (ROW*ROW) //2048*2048
#define THREADS_PER_BLOCK 1 //1024
#define RADIUS 3
#define BLOCK_SIZE (THREADS_PER_BLOCK-2*RADIUS)

// forward declaration
__global__ void td(int *in, int *out);


void random_ints(int * mat, int n) {
  srand(time(0));
  int i;
  for (i = 0; i < n; i++) {
    mat[i] = (rand() % 2) * 10;
  }
}

void printMatrix(int * mat, int n) {

  int i, j;
  for (i = 0; i < ROW; i++) {
    for (j = 0; j < ROW; j++) {
      printf("%d ", mat[i * ROW + j]);
    }
    printf("\n");
  }

}


int main(void) {
  int *in, *out;     // host copies a,b
  int *d_in,*d_out;  // device copies a,b
  int size = N * sizeof(int);

  // Alloc space for host copies a,b and setup input
  in  = (int *)malloc(size);

  random_ints(in, N);
  out = (int *)malloc(size);

  printf("IN\n");
  printMatrix(in, N);

  // Allocate space for device copies of in, out
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);

  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

  td<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_in,d_out);


  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

  printf("OUT\n");
  printMatrix(out, N);

  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  return 0;
}



__global__ void td(int *in, int *out) {
  int first_index = 0;
  int current_index = threadIdx.x + blockIdx.x * blockDim.x;
  int last_index = (ROW * ROW); // dimensao do bloco * numero_de_blocos + threads por bloco

  int distancia_esq = 0;
  int distancia_dir = 0;
  // olha todas as casas à esquerda da posição original
  for(int offset = current_index; offset >= 0; offset--) {
	if((int)in[offset] == 0){
		// achou a distancia minima a esquerda
		out[current_index] = distancia_esq;
		break;
	}
	// nao achou incrementa e continua
	distancia_esq++;
  }
  __syncthreads(); // espera todos à esquerda terminar

  // olha todas as casas à direita da posição original
  for(int offset = current_index; offset < last_index; offset++){
	if((int)in[offset] == 0) {
		// ver se a distancia a direita eh menor que a esquerda ja calculada
		if(out[current_index] > distancia_dir) {
			out[current_index] = distancia_dir;
		}
		break; // achou a distancia minima a direita

	}
	distancia_dir++;
  }



  __syncthreads(); // espera todas à direita terminar
}
