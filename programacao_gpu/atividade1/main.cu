
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

int* m_malloc(int n) {
  return (int* )malloc(sizeof (int) * n * n);
}
void m_fill_random(int * mat, int n) {
  //srand((unsigned int)time(NULL));
  // int seed = rand();

  // use same seed to draw the same image again on every test
  srand(rand());
  int i;
  for (i = 0; i < n * n; i++) {
    mat[i] = rand() % 10;
  }
}

void mm_cpu(int * A, int * B, int * C, int n) {
  int i, j, k;
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++) {
      C[i * n + j] = 0;
      for (k = 0; k < n; k++)
        C[i * n + j] += A[i * n + k] * B[k * n + j];
    }
}

void print_mat(int * A, int * B, int * C, int n) {
  int i, j;
  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++)
      printf("%d ", A[i * n + j]);
    printf("\n");
  }
  printf("\n\n");

  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++)
      printf("%d ", B[i * n + j]);
    printf("\n");
  }
  printf("\n \n");

  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++)
      printf("%d ", C[i * n + j]);
    printf("\n");
  }
}

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
  int width;
  int height;
  int stride; // necessario para o metodo com shared memory
  int* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// GET e SET elementos do device

__device__ float GetElement(const Matrix A, int row, int col) {  // Get a matrix element
    return A.elements[row * A.stride + col];
}

__device__ void SetElement(Matrix A, int row, int col, float value) {  // Set a matrix element
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is located col sub-matrices to the right and row 
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col)    {        // sub-matrices down from the upper-left corner of A
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return Asub;
}

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel_g(const Matrix, const Matrix, Matrix);
__global__ void MatMulKernel_s(const Matrix, const Matrix, Matrix);
void MatMul_g(const Matrix A, const Matrix B, Matrix C, int device);
void MatMul_s(const Matrix A, const Matrix B, Matrix C, int device);

int main() {

  clock_t t;
  float time;
  int co = 2;
  int size[7] = {256, 512, 1024, 2048, 4096, 8192, 16384}; // more sizes

	int n = size[co];

	printf("======= Matriz %dx%d =======\n", n, n);

	int* A = m_malloc(n);
	int* B = m_malloc(n);
	int* C = m_malloc(n);
	
	Matrix mA;
	mA.width = n;
	mA.height = n;
	mA.elements = A;

	Matrix mB;
	mB.width = n;
	mB.height = n;
	mB.elements = B;

	Matrix mC;
	mC.width = n;
	mC.height = n;
	mC.elements = C;

	m_fill_random(A, n);
	m_fill_random(B, n);

	  time = 0;
	  t = clock();
	  mm_cpu(A, B, C, n);
	  time = (float)(clock() - t);
	  time = time / CLOCKS_PER_SEC;
	  printf("CPU   = %10.1fms \n", time * 1000);

	  int count = 1;

	  hipDeviceProp_t prop;
   	  hipGetDeviceProperties(&prop, count);
	  hipSetDevice(count);

	  t = clock();
	  MatMul_g(mA, mB, mC, count);
	  time = (float)(clock() - t);
	  time = time / CLOCKS_PER_SEC;
	  printf("GPU_GLOBAL - %s = %10.1fms \n", prop.name, time * 1000);

	  t = clock();
	  MatMul_s(mA, mB, mC, count);
	  time = (float)(clock() - t);
	  time = time / CLOCKS_PER_SEC;
	  printf("GPU_SHARED - %s = %10.1fms \n", prop.name, time * 1000);

	  free(A);
	  free(B);
	  free(C);
  return 0;
}



// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul_g(const Matrix A, const Matrix B, Matrix C, int device) {
  // Load A and B to device memory
  Matrix d_A;
  d_A.width = A.width;
  d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipSetDevice(device);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width = B.width;
  d_B.height = B.height;
  size = B.width * B.height * sizeof(float);
  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  // Allocate C in device memory
  Matrix d_C;
  d_C.width = C.width;
  d_C.height = C.height;
  size = C.width * C.height * sizeof(float);
  hipMalloc(&d_C.elements, size);

  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
  MatMulKernel_g <<< dimGrid, dimBlock>>>(d_A, d_B, d_C);

  // Read C from device memory
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}

// PRATICAMENTE IGUAL AO ANTERIOR, COM A DIFERENCA
// QUE O KERNEL INVOCADO AQUI EH O DE SHARED AO INVES DO GLOBAL
void MatMul_s(const Matrix A, const Matrix B, Matrix C, int device) {
  // Load A and B to device memory
  Matrix d_A;
  d_A.width = A.width;
  d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipSetDevice(device);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width = B.width;
  d_B.height = B.height;
  size = B.width * B.height * sizeof(float);
  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  // Allocate C in device memory
  Matrix d_C;
  d_C.width = C.width;
  d_C.height = C.height;
  size = C.width * C.height * sizeof(float);
  hipMalloc(&d_C.elements, size);

  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
  MatMulKernel_s <<< dimGrid, dimBlock>>>(d_A, d_B, d_C);

  // Read C from device memory
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}

// MULTIPLICACAO DE MATRIZES UTILIZANDO MEMORIA GLOBAL
__global__ void MatMulKernel_g(Matrix A, Matrix B, Matrix C) {
  // Each thread computes one element of C by accumulating results into Cvalue
  float Cvalue = 0;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  for (int e = 0; e < A.width; ++e)
    Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
  C.elements[row * C.width + col] = Cvalue;
}

// MULTIPLICACAO DE MATRIZES UTILIZANDO MEMORIA COMPARTILHADA
__global__ void MatMulKernel_s(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;


    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
	for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
		// Get sub-matrix Asub of A
		Matrix Asub = GetSubMatrix(A, blockRow, m);
		// Get sub-matrix Bsub of B
		Matrix Bsub = GetSubMatrix(B, m, blockCol);
		// Shared memory used to store Asub and Bsub respectively
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
		// Load Asub and Bsub from device memory to shared memory
		// Each thread loads one element of each sub-matrix
		As[row][col] = GetElement(Asub, row, col);
		Bs[row][col] = GetElement(Bsub, row, col);
		// Synchronize to make sure the sub-matrices are loaded
		// before starting the computation
		__syncthreads();
	       // Multiply Asub and Bsub together
		for (int e = 0; e < BLOCK_SIZE; ++e)
		    Cvalue += As[row][e] * Bs[e][col];
		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	    }
	    // Write Csub to device memory Each thread writes one element
	    SetElement(Csub, row, col, Cvalue);
}
